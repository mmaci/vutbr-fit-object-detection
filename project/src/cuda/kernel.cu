#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <opencv2/core/core.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/features2d/features2d.hpp>
#include <opencv2/nonfree/features2d.hpp>

#include <iostream>
#include <vector>
#include <fstream>
#include <stdio.h>

#include "header.h"
#include "detector.h"
#include "alphas.h"

#include <algorithm>
#include <vector>

/// wrapper to call kernels
hipError_t runKernelWrapper(uint8* /* device image */, Detection* /* device detection buffer */, uint32* /* device detection count */, SurvivorData*, Bounds*, const DetectorInfo);

/// runs object detectin on gpu itself
__device__ void detectSurvivors(uint8*, SurvivorData*, uint16, uint16);
__device__ void detectSurvivorsInit(uint8*, SurvivorData*, uint16);
__device__ void detectDetections(uint8*, SurvivorData*, Detection*, uint32*, uint16, Bounds*);
/// gpu bilinear interpolation
__device__ void bilinearInterpolation(uint8* /* output image */, const float /* scale */);
/// builds a pyramid image with parameters set in header.h
__device__ void buildPyramid(uint8* /* device image */, uint32, uint32, uint32, uint32, Bounds*, uint32, uint32);

/// detector stages
__constant__ Stage stages[STAGE_COUNT];
/// detector parameters
__constant__ DetectorInfo detectorInfo[1];

/// pyramid kernel

texture<uint8> textureOriginalImage;
texture<uint8> texturePyramidImage;
texture<float> textureAlphas;

uint32 param = OPT_ALL;

__global__ void pyramidImageKernel(uint8* imageData, Bounds* bounds)
{
	buildPyramid(imageData, 320, 240, 48, 48, bounds, 8, 4);
}

__device__ void buildPyramid(uint8* imageData, uint32 max_x, uint32 max_y, uint32 min_x, uint32 min_y, Bounds* bounds, uint32 octaves, uint32 levels_per_octave)
{
	// coords in the original image
	const int threadId = (blockIdx.x * blockDim.x) + threadIdx.x;
	const int x = threadId % DET_INFO.pyramidImageWidth;
	const int y = threadId / DET_INFO.pyramidImageWidth;

	// only index data in the original image
	if (x < (DET_INFO.imageWidth - 1) && y < (DET_INFO.imageHeight - 1))
	{

		float scaling_factor = pow(2.0f, 1.0f / levels_per_octave);
		bool is_landscape = DET_INFO.imageWidth > DET_INFO.imageHeight;

		uint32 init_offset = DET_INFO.pyramidImageWidth * DET_INFO.imageHeight;
		uint32 init_y_offset = DET_INFO.imageHeight;
		uint32 init_x_offset = 0;

		uint32 offset, y_offset = init_y_offset, x_offset;
		for (uint8 octave = 0; octave < octaves; ++octave)
		{
			uint32 max_width = max_x / (octave + 1);
			uint32 max_height = max_y / (octave + 1);

			// box to which fit the resized image
			float current_scale = is_landscape ? (float)DET_INFO.imageWidth / (float)max_width : (float)DET_INFO.imageHeight / (float)max_height;

			uint32 image_width = DET_INFO.imageWidth / current_scale;
			uint32 image_height = DET_INFO.imageHeight / current_scale;

			// set current X-offset to the beginning and total offset based on current octave
			x_offset = init_x_offset;
			offset = init_offset;
			for (uint8 i = 0; i < octave; ++i)
				offset += (max_y / (i + 1)) * DET_INFO.pyramidImageWidth;

			// set starting scale based on current octave		
			uint32 final_y_offset = image_height;

			// process all levels of the pyramid
			for (uint8 level = 0; level < levels_per_octave; ++level)
			{
				bilinearInterpolation(imageData + offset, current_scale);

				if (x == 0 && y == 0) {
					uint32 bounds_id = levels_per_octave * octave + level;
					bounds[bounds_id].offset = offset;
					bounds[bounds_id].y_offset = y_offset;
					bounds[bounds_id].x_offset = x_offset;
					bounds[bounds_id].width = image_width;
					bounds[bounds_id].height = image_height;
					bounds[bounds_id].scale = current_scale;
				}

				current_scale *= scaling_factor;
				x_offset += image_width;
				offset += image_width;

				image_width = (float)DET_INFO.imageWidth / current_scale;
				image_height = (float)DET_INFO.imageHeight / current_scale;

				if (image_width < min_x || image_height < min_y)
					break;
			}

			y_offset += final_y_offset;
		}
	}
}

/** @brief Kernel wrapper around detection processing, outputting detections.
* @see detectDetections
*
* @param imageData			Input image.
* @param detections			Ouptut array of detections.
* @param detectionCount		Output number of detections.
* @param survivors			Initial array of threads, which still process the detection.
* @param survivorCount		Initial number of threads, which still process the detection.
* @param bounds				Data about the different subsampled images.
* @return Void.
*/
__global__ void detectionKernel(
	uint8*			imageData,
	Detection*		detections,
	uint32*			detectionCount,
	SurvivorData*	survivors,
	Bounds*			bounds)
{	
	detectSurvivorsInit(imageData, survivors, 16);
	detectSurvivors(imageData, survivors, 16, 32);
	detectSurvivors(imageData, survivors, 32, 64);
	detectSurvivors(imageData, survivors, 64, 128);
	detectSurvivors(imageData, survivors, 128, 256);
	detectSurvivors(imageData, survivors, 256, 512);		
	detectDetections(imageData, survivors, detections, detectionCount, 512, bounds);
}

__device__ void bilinearInterpolation(uint8* outImage, float scale)
{
	const int threadId = (blockIdx.x * blockDim.x) + threadIdx.x;
	const int origX = threadId % DET_INFO.pyramidImageWidth;
	const int origY = threadId / DET_INFO.pyramidImageWidth;

	const int x = (float)origX / scale;
	const int y = (float)origY / scale;

	uint8 res = tex1Dfetch(textureOriginalImage, origY * DET_INFO.imageWidth + origX);

	outImage[y * DET_INFO.pyramidImageWidth + x] = res;
}

__device__ void sumRegions(uint8* imageData, uint32 x, uint32 y, Stage* stage, uint32* values)
{
	values[0] = tex1Dfetch(texturePyramidImage, y * DET_INFO.imageWidth + x);
	x += stage->width;
	values[1] = tex1Dfetch(texturePyramidImage, y * DET_INFO.imageWidth + x);
	x += stage->width;
	values[2] = tex1Dfetch(texturePyramidImage, y * DET_INFO.imageWidth + x);
	y += stage->height;
	values[5] = tex1Dfetch(texturePyramidImage, y * DET_INFO.imageWidth + x);
	y += stage->height;
	values[8] = tex1Dfetch(texturePyramidImage, y * DET_INFO.imageWidth + x);
	x -= stage->width;
	values[7] = tex1Dfetch(texturePyramidImage, y * DET_INFO.imageWidth + x);
	x -= stage->width;
	values[6] = tex1Dfetch(texturePyramidImage, y * DET_INFO.imageWidth + x);
	y -= stage->height;
	values[3] = tex1Dfetch(texturePyramidImage, y * DET_INFO.imageWidth + x);
	x += stage->width;
	values[4] = tex1Dfetch(texturePyramidImage, y * DET_INFO.imageWidth + x);
}

__device__ float evalLBP(uint8* data, uint32 x, uint32 y, Stage* stage)
{
	const uint8 LBPOrder[8] = { 0, 1, 2, 5, 8, 7, 6, 3 };

	uint32 values[9];

	sumRegions(data, x, y, stage, values);

	uint8 code = 0;
	for (uint8 i = 0; i < 8; ++i)
		code |= (values[LBPOrder[i]] > values[4]) << i;

	return tex1Dfetch(textureAlphas, stage->alphaOffset + code);
}

__device__ bool eval(uint8* imageData, uint32 x, uint32 y, float* response, uint16 startStage, uint16 endStage)
{
	for (uint16 i = startStage; i < endStage; ++i) {
		Stage stage = stages[i];
		*response += evalLBP(imageData, x + stage.x, y + stage.y, &stage);
		if (*response < stage.thetaB) {
			return false;
		}
	}

	// final waldboost threshold
	return *response > FINAL_THRESHOLD;
}

/** @brief Final detection processing
*
* Processes detections on an image beginning at a starting stage, until the end.
* Processes only given surviving positions and outputs detections, which can then
* be displayed.
*
* @param imageData			Input image.
* @param survivors			Input array of surviving positions.
* @param detections			Output array of detections.
* @param detectionCount		Number of detections.
* @param startStage			Starting stage of the waldboost detector.
* @param bounds				Data about the different subsampled images.
* @return Void.
*/
__device__ void detectDetections(
	uint8*			imageData, 
	SurvivorData*	survivors,
	Detection*		detections, 
	uint32*			detectionCount, 
	uint16			startStage,	
	Bounds*			bounds)
{	
	const int globalId = blockIdx.x*blockDim.x + threadIdx.x;

	if (globalId < (DET_INFO.pyramidImageWidth - DET_INFO.classifierWidth) * (DET_INFO.pyramidImageHeight - DET_INFO.classifierHeight)) {

		float response = survivors[globalId].response;

		if (response < FINAL_THRESHOLD)
			return;

		const uint32 x = survivors[globalId].x;
		const uint32 y = survivors[globalId].y;

		__syncthreads();

		bool survived = eval(imageData, x, y, &response, startStage, STAGE_COUNT);
		if (survived) {
			Bounds b;
			for (uint8 i = 0; i < 8 * 3; ++i) {
				if (x >= bounds[i].x_offset && x < (bounds[i].x_offset + bounds[i].width) &&
					y >= bounds[i].y_offset && y < (bounds[i].y_offset + bounds[i].height)) {
					b = bounds[i];
					break;
				}
			}

			uint32 pos = atomicInc(detectionCount, 2048);
			detections[pos].x = (float)(x - b.x_offset) * b.scale;
			detections[pos].y = (float)(y - b.y_offset) * b.scale;
			detections[pos].width = DET_INFO.classifierWidth * b.scale;
			detections[pos].height = DET_INFO.classifierHeight * b.scale;
			detections[pos].response = response;
		}
	}
}
/** @brief Initial survivor detection processing
 *
 * Processes detections on an image from the first stage (of the waldboost detector). 
 * Processes the whole image and outputs the remaining surviving positions after reaching 
 * the ending stage.
 *
 * @param imageData			Input image.
 * @param survivors			Output array of surviving positions. 
 * @param endStage			Ending stage of the waldboost detector.
 * @return Void.
 *
 * @todo calculate newThreadId using prefix sum and shared memory to remove global memory 
 *		atomic instructio bottlenect
 */
__device__ void detectSurvivorsInit(
	uint8*			imageData,
	SurvivorData*	survivors,
	uint16			endStage)
{
	__shared__ uint32 localSurvivors[BLOCK_SIZE];

	const int threadId = threadIdx.x;
	const int globalId = blockIdx.x*blockDim.x + threadIdx.x;

	if (globalId < (DET_INFO.pyramidImageWidth - DET_INFO.classifierWidth) * (DET_INFO.pyramidImageHeight - DET_INFO.classifierHeight)) {

		const int x = globalId % (DET_INFO.pyramidImageWidth - DET_INFO.classifierWidth);
		const int y = globalId / (DET_INFO.pyramidImageWidth - DET_INFO.classifierWidth);
		
		float response = 0.0f;
		bool survived = eval(imageData, x, y, &response, 0, endStage);

		localSurvivors[threadId] = static_cast<uint32>(survived);

		// up-sweep
		int offset = 1;
		for (uint32 d = BLOCK_SIZE >> 1; d > 0; d >>= 1, offset <<= 1) {
			__syncthreads();

			if (threadId < d) {
				uint32 ai = offset * (2 * threadId + 1) - 1;
				uint32 bi = offset * (2 * threadId + 2) - 1;
				localSurvivors[bi] += localSurvivors[ai];
			}
		}

		// down-sweep
		if (threadId == 0) {
			localSurvivors[BLOCK_SIZE - 1] = 0;
		}

		for (uint32 d = 1; d < BLOCK_SIZE; d <<= 1) {
			offset >>= 1;

			__syncthreads();

			if (threadId < d) {
				uint32 ai = offset * (2 * threadId + 1) - 1;
				uint32 bi = offset * (2 * threadId + 2) - 1;

				uint32 t = localSurvivors[ai];
				localSurvivors[ai] = localSurvivors[bi];
				localSurvivors[bi] += t;
			}
		}
		
		survivors[globalId].response = BAD_RESPONSE;

		__syncthreads();
		
		if (survived) {									
			uint32 newThreadId = blockIdx.x*blockDim.x + localSurvivors[threadId];
			// save position and current response
			survivors[newThreadId].x = x;
			survivors[newThreadId].y = y;
			survivors[newThreadId].response = response;
		}		
	}
}

/** @brief Survivor detection processing
*
* Processes detections on an image from a set starting stage (of the waldboost detector).
* Processes only positions in the initSurvivors array and outputs still surviving positions
* after reaching the ending stage.
*
* @param imageData			Input image.
* @param survivors			Output and input array of surviving positions.
* @param startStage			Starting stage of the waldboost detector.
* @param endStage			Ending stage of the waldboost detector.
* @return Void.
*
* @todo calculate newThreadId using prefix sum and shared memory to remove global memory
*		atomic instructio bottlenect
*/
__device__ void detectSurvivors(
	uint8*			imageData, 
	SurvivorData*	survivors,
	uint16			startStage, 
	uint16			endStage)								
{
	__shared__ uint32 localSurvivors[BLOCK_SIZE];

	const int threadId = threadIdx.x;
	const int globalId = blockIdx.x*blockDim.x + threadIdx.x;

	if (globalId < (DET_INFO.pyramidImageWidth - DET_INFO.classifierWidth) * (DET_INFO.pyramidImageHeight - DET_INFO.classifierHeight)) {
		
		float response = survivors[globalId].response;
		
		if (response < FINAL_THRESHOLD)
			return;

		const uint32 x = survivors[globalId].x;
		const uint32 y = survivors[globalId].y;

		__syncthreads();

		bool survived = eval(imageData, x, y, &response, startStage, endStage);
		
		localSurvivors[threadId] = static_cast<uint32>(survived);

		// up-sweep
		int offset = 1;
		for (uint32 d = BLOCK_SIZE >> 1; d > 0; d >>= 1, offset <<= 1) {
			__syncthreads();

			if (threadId < d) {
				uint32 ai = offset * (2 * threadId + 1) - 1;
				uint32 bi = offset * (2 * threadId + 2) - 1;
				localSurvivors[bi] += localSurvivors[ai];
			}
		}

		// down-sweep
		if (threadId == 0) {
			localSurvivors[BLOCK_SIZE - 1] = 0;
		}

		for (uint32 d = 1; d < BLOCK_SIZE; d <<= 1) {
			offset >>= 1;

			__syncthreads();

			if (threadId < d) {
				uint32 ai = offset * (2 * threadId + 1) - 1;
				uint32 bi = offset * (2 * threadId + 2) - 1;

				uint32 t = localSurvivors[ai];
				localSurvivors[ai] = localSurvivors[bi];
				localSurvivors[bi] += t;
			}
		}

		survivors[globalId].response = BAD_RESPONSE;

		__syncthreads();

		if (survived) {
			uint32 newThreadId = blockIdx.x*blockDim.x + localSurvivors[threadId];
			// save position and current response
			survivors[newThreadId].x = x;
			survivors[newThreadId].y = y;
			survivors[newThreadId].response = response;
		}
	}
}

hipError_t runKernelWrapper(
	uint8* imageData, 
	Detection* detections, 
	uint32* detectionCount, 
	SurvivorData* survivors,
	Bounds* bounds, 
	const DetectorInfo info)
{
	hipEvent_t start_detection, stop_detection, start_pyramid, stop_pyramid;
	hipEventCreate(&start_detection);
	hipEventCreate(&stop_detection);
	hipEventCreate(&start_pyramid);
	hipEventCreate(&stop_pyramid);

	float pyramid_time = 0.f, detection_time = 0.f;

	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);

	dim3 grid(4096, 1, 1);
	dim3 block(1024, 1, 1);

	if (param & OPT_TIMER)
		hipEventRecord(start_pyramid);

	pyramidImageKernel <<<grid, block>>> (imageData, bounds);

	if (param & OPT_TIMER)
	{
		hipEventRecord(stop_pyramid);
		hipEventSynchronize(stop_pyramid);
		hipEventElapsedTime(&pyramid_time, start_pyramid, stop_pyramid);
		printf("PyramidKernel time: %f ms\n", pyramid_time);
	}

	hipDeviceSynchronize();

	// bind created pyramid to texture memory
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<uint8>();
	hipBindTexture(nullptr, &texturePyramidImage, imageData, &channelDesc, sizeof(uint8) * info.pyramidImageHeight * info.pyramidImageWidth);

	hipEventRecord(start_detection);

	detectionKernel <<<grid, block>>>(imageData, detections, detectionCount, survivors, bounds);

	hipUnbindTexture(texturePyramidImage);

	hipEventRecord(stop_detection);
	hipEventSynchronize(stop_detection);
	hipEventElapsedTime(&detection_time, start_detection, stop_detection);

	if (param & OPT_TIMER)
	{
		printf("DetectionKernel time: %f ms\n", detection_time);
		printf("Total time: %f ms \n", pyramid_time + detection_time);
	}

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		std::cerr << "[" << LIBNAME << "]: " << hipGetErrorString(cudaStatus) << std::endl;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		std::cerr << "[" << LIBNAME << "]: " << "hipDeviceSynchronize failed (error code: " << cudaStatus << ")" << std::endl;
	}

	return cudaStatus;
}

bool runDetector(cv::Mat* image)
{
	cv::Mat image_bw;

	// TODO: do b&w conversion on GPU
	cvtColor(*image, image_bw, CV_BGR2GRAY);

	// TODO: rewrite this
	const size_t ORIG_IMAGE_SIZE = image_bw.cols * image_bw.rows * sizeof(uint8);
	const size_t PYRAMID_IMAGE_HEIGHT = image_bw.rows * 3;
	const size_t PYRAMID_IMAGE_WIDTH = image_bw.cols;
	const size_t PYRAMID_IMAGE_SIZE = PYRAMID_IMAGE_HEIGHT * PYRAMID_IMAGE_WIDTH;


	// ********* DEVICE VARIABLES **********
	float* devAlphaBuffer;
	uint8* devImageData, *devOriginalImage;
	uint32* devDetectionCount;
	Detection* devDetections;
	Bounds* devBounds;
	SurvivorData* devSurvivors;

	// ********* HOST VARIABLES *********
	uint8* hostImageData;
	hostImageData = (uint8*)malloc(sizeof(uint8) * PYRAMID_IMAGE_SIZE);
	uint32 hostDetectionCount = 0;
	Detection hostDetections[MAX_DETECTIONS];

	// ********* CONSTANTS **********
	DetectorInfo hostDetectorInfo[1];
	hostDetectorInfo[0].imageWidth = image_bw.cols;
	hostDetectorInfo[0].imageHeight = image_bw.rows;
	hostDetectorInfo[0].pyramidImageWidth = PYRAMID_IMAGE_WIDTH;
	hostDetectorInfo[0].pyramidImageHeight = PYRAMID_IMAGE_HEIGHT;
	hostDetectorInfo[0].classifierWidth = CLASSIFIER_WIDTH;
	hostDetectorInfo[0].classifierHeight = CLASSIFIER_HEIGHT;
	hostDetectorInfo[0].alphaCount = ALPHA_COUNT;
	hostDetectorInfo[0].stageCount = STAGE_COUNT;

	// ********* GPU MEMORY ALLOCATION-COPY **********		
	// constant memory
	hipMemcpyToSymbol(HIP_SYMBOL(stages), hostStages, sizeof(Stage) * STAGE_COUNT);
	hipMemcpyToSymbol(HIP_SYMBOL(detectorInfo), hostDetectorInfo, sizeof(DetectorInfo));

	// texture memory		
	hipMalloc(&devAlphaBuffer, STAGE_COUNT * ALPHA_COUNT * sizeof(float));
	hipMemcpy(devAlphaBuffer, alphas, STAGE_COUNT * ALPHA_COUNT * sizeof(float), hipMemcpyHostToDevice);

	hipMalloc((void**)&devImageData, PYRAMID_IMAGE_SIZE * sizeof(uint8));
	hipMalloc((void**)&devOriginalImage, ORIG_IMAGE_SIZE * sizeof(uint8));
	hipMalloc((void**)&devDetectionCount, sizeof(uint32));
	hipMalloc((void**)&devDetections, MAX_DETECTIONS * sizeof(Detection));
	hipMalloc((void**)&devBounds, PYRAMID_IMAGE_COUNT * sizeof(Bounds));
	hipMalloc((void**)&devSurvivors, PYRAMID_IMAGE_SIZE * sizeof(SurvivorData));	

	uint8* clean = (uint8*)malloc(PYRAMID_IMAGE_SIZE * sizeof(uint8));
	memset(clean, 0, PYRAMID_IMAGE_SIZE * sizeof(uint8));
	hipMemcpy(devImageData, clean, PYRAMID_IMAGE_SIZE * sizeof(uint8), hipMemcpyHostToDevice);
	free(clean);

	hipMemcpy(devImageData, image_bw.data, ORIG_IMAGE_SIZE * sizeof(uint8), hipMemcpyHostToDevice);
	hipMemcpy(devOriginalImage, image_bw.data, ORIG_IMAGE_SIZE * sizeof(uint8), hipMemcpyHostToDevice);
	hipMemcpy(devDetectionCount, &hostDetectionCount, sizeof(uint32), hipMemcpyHostToDevice);

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<uint8>();
	hipBindTexture(nullptr, &textureOriginalImage, devOriginalImage, &channelDesc, sizeof(uint8) * ORIG_IMAGE_SIZE);

	hipChannelFormatDesc alphaChannelDesc = hipCreateChannelDesc<float>();
	hipBindTexture(nullptr, &textureAlphas, devAlphaBuffer, &alphaChannelDesc, STAGE_COUNT * ALPHA_COUNT * sizeof(float));

	// ********* RUN ALL THEM KERNELS! **********		

	hipError_t cudaStatus = runKernelWrapper(
		devImageData,
		devDetections,
		devDetectionCount,
		devSurvivors,		
		devBounds,
		hostDetectorInfo[0]
		);

	// ********* COPY RESULTS FROM GPU *********

	hipMemcpy(&hostDetectionCount, devDetectionCount, sizeof(uint32), hipMemcpyDeviceToHost);
	hipMemcpy(hostDetections, devDetections, hostDetectionCount * sizeof(Detection), hipMemcpyDeviceToHost);
	hipMemcpy(hostImageData, devImageData, sizeof(uint8) * PYRAMID_IMAGE_SIZE, hipMemcpyDeviceToHost);

	// ********* FREE CUDA MEMORY *********
	hipUnbindTexture(textureOriginalImage);
	hipUnbindTexture(textureAlphas);

	hipFree(devImageData);
	hipFree(devOriginalImage);
	hipFree(devDetections);
	hipFree(devDetectionCount);
	hipFree(devAlphaBuffer);
	hipFree(devBounds);
	hipFree(devSurvivors);

	// ********* SHOW RESULTS *********	

	if (param & OPT_VERBOSE)
		std::cout << "Detection count: " << hostDetectionCount << std::endl;
	
	for (uint32 i = 0; i < hostDetectionCount; ++i)
	{
		if (param & OPT_VERBOSE)
			std::cout << "[" << hostDetections[i].x << "," << hostDetections[i].y << "," << hostDetections[i].width << "," << hostDetections[i].height << "] " << hostDetections[i].response << ", ";

		if (param & OPT_VISUAL_OUTPUT)
			cv::rectangle(*image, cvPoint(hostDetections[i].x, hostDetections[i].y), cvPoint(hostDetections[i].x + hostDetections[i].width, hostDetections[i].y + hostDetections[i].height), CV_RGB(0, 255, 0), 1);
	}

	// ******** FREE HOST MEMORY *********
	free(hostImageData);

	if (cudaStatus != hipSuccess) {
		std::cerr << "[" << LIBNAME << "]: " << "CUDA runtime error" << std::endl;;
		return false;
	}

	// needed for profiling - NSight
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		std::cerr << "[" << LIBNAME << "]: " << "hipDeviceReset failed" << std::endl;;
		return false;
	}

	return true;
}

/**
 * @todo move stuff here from runDetector
 */
void initDetector() {

}

/**
* @todo move stuff here from runDetector
*/
void freeDetector() {

}

bool process(std::string inFilename, Filetypes inFileType) 
{
	cv::Mat image;
	switch (inFileType)
	{
		case INPUT_IMAGE:
		{
			image = cv::imread(inFilename.c_str(), CV_LOAD_IMAGE_COLOR);

			if (!image.data)
				std::cerr << "[" << LIBNAME << "]: " << "Could not open or find the image (filename: " << inFilename << ")" << std::endl;

			runDetector(&image);

			if (param & OPT_VISUAL_OUTPUT)
			{
				cv::imshow(LIBNAME, image);
				cv::waitKey(WAIT_DELAY);
			}

			break;
		}
		case INPUT_DATASET:
		{
			std::ifstream in;
			in.open(inFilename);
			std::string file;
			while (!in.eof())
			{
				std::getline(in, file);
				image = cv::imread(file.c_str(), CV_LOAD_IMAGE_COLOR);

				if (!image.data)
				{
					std::cerr << "[" << LIBNAME << "]: " << "Could not open or find the image (inFilename: " << file.c_str() << ")" << std::endl;
					continue;
				}

				runDetector(&image);

				if (param & OPT_VISUAL_OUTPUT)
				{
					cv::imshow(LIBNAME, image);
					cv::waitKey(WAIT_DELAY);
				}
			}
			break;
		}
		case INPUT_VIDEO:
		{
			cv::VideoCapture video;

			video.open(inFilename);
			initDetector();
			while (true) {				
				video >> image;

				if (image.empty())
					break;

				runDetector(&image);

				if (param & OPT_VISUAL_OUTPUT)
				{
					cv::imshow(LIBNAME, image);
					cv::waitKey(WAIT_DELAY);
				}
			}
			freeDetector();
			video.release();
			break;
		}
		default:
			return false;
	}

	return true;
}

int main(int argc, char** argv)
{
	std::string inputFilename;
	Filetypes mode;
	for (int i = 1; i < argc; ++i)
	{
		if (std::string(argv[i]) == "-ii" && i + 1 < argc) {
			mode = INPUT_IMAGE;
			inputFilename = argv[++i];
		}
		else if (std::string(argv[i]) == "-di" && i + 1 < argc) {
			mode = INPUT_DATASET;
			inputFilename = argv[++i];
		}
		else if (std::string(argv[i]) == "-iv" && i + 1 < argc) {
			mode = INPUT_VIDEO;
			inputFilename = argv[++i];
		}		
		else {
			std::cerr << "Usage: " << argv[0] << " -ii [input file] or -di [dataset] or -iv [input video]" << std::endl;
			return EXIT_FAILURE;
		}
	}

	process(inputFilename, mode);

	return EXIT_SUCCESS;
}
